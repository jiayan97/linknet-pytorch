#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "roi_align_kernel.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
            i += blockDim.x * gridDim.x)


    __global__ void ROIAlignForward(const int nthreads, const float* image_ptr, const float* boxes_ptr,
         int num_boxes, int batch, int image_height, int image_width, int crop_height,
         int crop_width, int depth, float extrapolation_value, float* crops_ptr) {
    CUDA_1D_KERNEL_LOOP(out_idx, nthreads) {
        // (n, c, ph, pw) is an element in the aligned output
        int idx = out_idx;
        const int x = idx % crop_width;
        idx /= crop_width;
        const int y = idx % crop_height;
        idx /= crop_height;
        const int d = idx % depth;
        const int b = idx / depth;

        const int b_in = int(boxes_ptr[b*5]);
        const float x1 = boxes_ptr[b * 5 + 1];
        const float y1 = boxes_ptr[b * 5 + 2];
        const float x2 = boxes_ptr[b * 5 + 3];
        const float y2 = boxes_ptr[b * 5 + 4];
        if (b_in < 0 || b_in >= batch) {
            continue;
        }

        const float height_scale =
            (crop_height > 1) ? (y2 - y1) * (image_height - 1) / (crop_height - 1)
                              : 0;
        const float width_scale =
            (crop_width > 1) ? (x2 - x1) * (image_width - 1) / (crop_width - 1) : 0;

        const float in_y = (crop_height > 1)
                               ? y1 * (image_height - 1) + y * height_scale
                               : 0.5 * (y1 + y2) * (image_height - 1);
        if (in_y < 0 || in_y > image_height - 1) {
            crops_ptr[out_idx] = extrapolation_value;
            continue;
        }

        const float in_x = (crop_width > 1)
                               ? x1 * (image_width - 1) + x * width_scale
                               : 0.5 * (x1 + x2) * (image_width - 1);
        if (in_x < 0 || in_x > image_width - 1) {
          crops_ptr[out_idx] = extrapolation_value;
          continue;
        }

        const int top_y_index = floorf(in_y);
        const int bottom_y_index = ceilf(in_y);
        const float y_lerp = in_y - top_y_index;

        const int left_x_index = floorf(in_x);
        const int right_x_index = ceilf(in_x);
        const float x_lerp = in_x - left_x_index;

        const float top_left = image_ptr[((b_in*depth + d) * image_height
            + top_y_index) * image_width + left_x_index];
        const float top_right = image_ptr[((b_in*depth + d) * image_height
            + top_y_index) * image_width + right_x_index];
        const float bottom_left = image_ptr[((b_in*depth + d) * image_height
            + bottom_y_index) * image_width + left_x_index];
        const float bottom_right = image_ptr[((b_in*depth + d) * image_height
            + bottom_y_index) * image_width + right_x_index];

        const float top = top_left + (top_right - top_left) * x_lerp;
        const float bottom = bottom_left + (bottom_right - bottom_left) * x_lerp;
        crops_ptr[out_idx] = top + (bottom - top) * y_lerp;
        }
    }

    int ROIAlignForwardLaucher(const float* image_ptr, const float* boxes_ptr,
         int num_boxes,  int batch, int image_height, int image_width, int crop_height,
         int crop_width, int depth, float extrapolation_value, float* crops_ptr, hipStream_t stream) {

        const int kThreadsPerBlock = 1024;
        const int output_size = num_boxes * crop_height * crop_width * depth;
        hipError_t err;

        ROIAlignForward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>
        (output_size, image_ptr, boxes_ptr, num_boxes, batch, image_height, image_width,
         crop_height, crop_width, depth, extrapolation_value, crops_ptr);

        err = hipGetLastError();
        if(hipSuccess != err) {
            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }

        return 1;
    }

__global__ void ROIAlignBackward(
    const int nthreads, const float* grads_ptr, const float* boxes_ptr,
    int num_boxes, int batch, int image_height,
    int image_width, int crop_height, int crop_width, int depth,
    float* grads_image_ptr) {
  CUDA_1D_KERNEL_LOOP(out_idx, nthreads) {

    // out_idx = d + depth * (w + crop_width * (h + crop_height * b))
    int idx = out_idx;
    const int x = idx % crop_width;
    idx /= crop_width;
    const int y = idx % crop_height;
    idx /= crop_height;
    const int d = idx % depth;
    const int b = idx / depth;

    const int b_in = boxes_ptr[b * 5];
    const float x1 = boxes_ptr[b * 5 + 1];
    const float y1 = boxes_ptr[b * 5 + 2];
    const float x2 = boxes_ptr[b * 5 + 3];
    const float y2 = boxes_ptr[b * 5 + 4];
    if (b_in < 0 || b_in >= batch) {
      continue;
    }

    const float height_scale =
        (crop_height > 1) ? (y2 - y1) * (image_height - 1) / (crop_height - 1)
                          : 0;
    const float width_scale =
        (crop_width > 1) ? (x2 - x1) * (image_width - 1) / (crop_width - 1) : 0;

    const float in_y = (crop_height > 1)
                           ? y1 * (image_height - 1) + y * height_scale
                           : 0.5 * (y1 + y2) * (image_height - 1);
    if (in_y < 0 || in_y > image_height - 1) {
      continue;
    }

    const float in_x = (crop_width > 1)
                           ? x1 * (image_width - 1) + x * width_scale
                           : 0.5 * (x1 + x2) * (image_width - 1);
    if (in_x < 0 || in_x > image_width - 1) {
      continue;
    }

    const int top_y_index = floorf(in_y);
    const int bottom_y_index = ceilf(in_y);
    const float y_lerp = in_y - top_y_index;

    const int left_x_index = floorf(in_x);
    const int right_x_index = ceilf(in_x);
    const float x_lerp = in_x - left_x_index;

    const float dtop = (1 - y_lerp) * grads_ptr[out_idx];
    atomicAdd(
        grads_image_ptr + ((b_in*depth + d)*image_height + top_y_index) * image_width + left_x_index,
        (1 - x_lerp) * dtop);
    atomicAdd(grads_image_ptr +
                      ((b_in * depth + d)*image_height+top_y_index)*image_width + right_x_index,
                       x_lerp * dtop);

    const float dbottom = y_lerp * grads_ptr[out_idx];
    atomicAdd(grads_image_ptr + ((b_in*depth+d)*image_height+bottom_y_index)*image_width+left_x_index,
        (1 - x_lerp) * dbottom);
    atomicAdd(grads_image_ptr + ((b_in*depth+d)*image_height+bottom_y_index)*image_width+right_x_index,
        x_lerp * dbottom);
  }
}

int ROIAlignBackwardLaucher(const float* grads_ptr, const float* boxes_ptr, int num_boxes,
    int batch, int image_height, int image_width, int crop_height, int crop_width, int depth,
    float* grads_image_ptr, hipStream_t stream) {
        const int kThreadsPerBlock = 1024;
        const int output_size = num_boxes * crop_height * crop_width * depth;
        hipError_t err;

        ROIAlignBackward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>
        (output_size, grads_ptr, boxes_ptr, num_boxes, batch, image_height, image_width, crop_height,
        crop_width, depth, grads_image_ptr);

        err = hipGetLastError();
        if(hipSuccess != err) {
            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }

        return 1;
    }


#ifdef __cplusplus
}
#endif


